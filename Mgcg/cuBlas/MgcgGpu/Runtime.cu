#include<hip/hip_runtime_api.h>
#include<hipblas.h>
#include<hipsparse.h>

extern "C"
{
	__declspec(dllexport) int _stdcall GetDeviceCount()
	{
		int count;
		::hipGetDeviceCount(&count);

		return count;
	}

	__declspec(dllexport) void _stdcall SetDevice(const int deviceID)
	{		
		::hipSetDevice(deviceID);
	}

	__declspec(dllexport) hipblasHandle_t* _stdcall CreateBlas()
	{
		::hipblasHandle_t* handle = new hipblasHandle_t();
		::hipblasCreate(handle);

		return handle;
	}

	__declspec(dllexport) void _stdcall DestroyBlas(hipblasHandle_t* cublas)
	{		
		::hipblasDestroy(*cublas);
		delete cublas;
	}

	__declspec(dllexport) hipsparseHandle_t* _stdcall CreateSparse()
	{
		::hipsparseHandle_t* handle = new hipsparseHandle_t();
		::hipsparseCreate(handle);

		return handle;
	}

	__declspec(dllexport) void _stdcall DestroySparse(hipsparseHandle_t* cusparse)
	{		
		::hipsparseDestroy(*cusparse);
		delete cusparse;
	}

	__declspec(dllexport) hipsparseMatDescr_t* _stdcall CreateMatDescr()
	{
		::hipsparseMatDescr_t* matDescr = new hipsparseMatDescr_t();
		hipsparseCreateMatDescr(matDescr);
		hipsparseSetMatType(*matDescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
		hipsparseSetMatIndexBase(*matDescr, HIPSPARSE_INDEX_BASE_ZERO);

		return matDescr;
	}

	__declspec(dllexport) void _stdcall DestroyMatDescr(hipsparseMatDescr_t* matDescr)
	{		
		::hipsparseDestroyMatDescr(*matDescr);
		delete matDescr;
	}
}