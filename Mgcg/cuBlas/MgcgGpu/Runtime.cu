#include<hip/hip_runtime_api.h>
#include<hipblas.h>
#include<hipsparse.h>

extern "C"
{
	__declspec(dllexport) int _stdcall GetDeviceCount()
	{
		int count;
		::hipGetDeviceCount(&count);

		return count;
	}

	__declspec(dllexport) hipblasHandle_t* _stdcall CreateBlas(const int deviceID)
	{
		hipSetDevice(deviceID);

		::hipblasHandle_t* handle = new hipblasHandle_t();
		::hipblasCreate(handle);

		return handle;
	}

	__declspec(dllexport) void _stdcall DestroyBlas(hipblasHandle_t* cublas, const int deviceID)
	{
		hipSetDevice(deviceID);
		
		::hipblasDestroy(*cublas);
		delete cublas;
	}

	__declspec(dllexport) hipsparseHandle_t* _stdcall CreateSparse(const int deviceID)
	{
		hipSetDevice(deviceID);

		::hipsparseHandle_t* handle = new hipsparseHandle_t();
		::hipsparseCreate(handle);

		return handle;
	}

	__declspec(dllexport) void _stdcall DestroySparse(hipsparseHandle_t* cusparse, const int deviceID)
	{
		hipSetDevice(deviceID);
		
		::hipsparseDestroy(*cusparse);
		delete cusparse;
	}

	__declspec(dllexport) hipsparseMatDescr_t* _stdcall CreateMatDescr(const int deviceID)
	{
		hipSetDevice(deviceID);

		::hipsparseMatDescr_t* matDescr = new hipsparseMatDescr_t();
		hipsparseCreateMatDescr(matDescr);
		hipsparseSetMatType(*matDescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
		hipsparseSetMatIndexBase(*matDescr, HIPSPARSE_INDEX_BASE_ZERO);

		return matDescr;
	}

	__declspec(dllexport) void _stdcall DestroyMatDescr(hipsparseMatDescr_t* matDescr, const int deviceID)
	{
		hipSetDevice(deviceID);
		
		::hipsparseDestroyMatDescr(*matDescr);
		delete matDescr;
	}
}